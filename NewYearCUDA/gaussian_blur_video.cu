#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <chrono>

__global__ void matrixElementSum(float *matrix, float *result, int size) {
    auto tid = threadIdx.x + threadIdx.y * blockDim.x;
    auto stride = blockDim.x * blockDim.y;

    // Perform block-wise reduction
    for (auto i = tid; i < size * size; i += stride) {
        atomicAdd(result, matrix[i]);
    }
}

__global__ void matrixElementWiseDivision(float *matrix, const float *divisor, int size) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < size && y < size) {
        matrix[y * size + x] /= *divisor;
    }
}

// Define CUDA kernel for Gaussian kernel generation
__global__ void generateGaussianKernel(float *kernel, int size, float strength) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < size && y < size) {
        float center = static_cast<float>(size - 1) / 2;

        float value = static_cast<float>(
                              1.0f
                              /
                              (2.0f * M_PI * powf(strength, 2))
                      ) *
                      exp(
                              -(
                                      powf(static_cast<float>(x) - center, 2) +
                                      powf(static_cast<float>(y) - center, 2)
                              )
                              /
                              (2 * strength * strength)
                      );

        kernel[y * size + x] = value;  // Divide by strength to match Python implementation
    }
}

// Define CUDA kernel for 2D convolution
__global__ void convolution2D(const uchar *input, uchar *output, int height, int width, int channels,
                              const float *kernel, int kernelSize, int padSize) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    auto c = blockIdx.z * blockDim.z + threadIdx.z;

    if (x < width - kernelSize + 1 && y < height - kernelSize + 1 && c < channels) {

        if (c == 3) {
            output[(y * width + x) * channels + c] = input[(y * width + x) * channels + c];
            return;
        }

        float current_channel_result = 0.0f;

        for (int i = 0; i < kernelSize; ++i) {
            for (int j = 0; j < kernelSize; ++j) {
                int imageX = static_cast<int>(x) + i - padSize;
                int imageY = static_cast<int>(y) + j - padSize;

                if (imageX >= 0 && imageX < width && imageY >= 0 && imageY < height) {
                    current_channel_result += kernel[i * kernelSize + j] *
                                              static_cast<float>(input[(imageY * width + imageX) * channels + c]);
                }
            }
        }

        output[(y * width + x) * channels + c] = static_cast<uchar>(current_channel_result);
    }
}

// Function to apply Gaussian blur using CUDA
void applyGaussianBlurCUDA(const cv::Mat &input, cv::Mat &output, int kernelSize, float strength) {

    auto width = input.cols;
    auto height = input.rows;

    int channels = input.channels();

    int padSize = kernelSize / 2;

    uchar *d_input, *d_output;
    float *d_kernel;

    hipMalloc((void **) &d_input, input.total() * input.elemSize());
    hipMalloc((void **) &d_output, output.total() * output.elemSize());
    hipMalloc((void **) &d_kernel, kernelSize * kernelSize * sizeof(float));

    hipMemcpy(d_input, input.ptr(), input.total() * input.elemSize(), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the Gaussian kernel generation kernel
    generateGaussianKernel<<<gridSize, blockSize>>>(d_kernel, kernelSize, strength);
    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // calc matrix sum
    float *dev_result;
    hipMalloc((void **) &dev_result, sizeof(float));
    // Using 4x4 thread blocks for block-wise reduction
    dim3 atomAddBlockDim(4, 4);
    dim3 atomAddGridDim(1, 1);
    matrixElementSum<<<atomAddGridDim, atomAddBlockDim>>>(d_kernel, dev_result, kernelSize);
    hipDeviceSynchronize();

    matrixElementWiseDivision<<<gridSize, blockSize>>>(d_kernel, dev_result, kernelSize);
    hipDeviceSynchronize();
    hipFree(dev_result);

    // Launch the 2D convolution kernel
//    convolution2D<<<gridSize, blockSize>>>(
//            d_input, d_output, height, width, channels, d_kernel, kernelSize, padSize
//    );

//    dim3 threadsPerBlock(channels, 1, 1);
//    dim3 blocksPerGrid((height - kernelSize + 1 + threadsPerBlock.x - 1) / threadsPerBlock.x,
//                       (width - kernelSize + 1 + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
//
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((width - kernelSize + 1 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height - kernelSize + 1 + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       (channels + threadsPerBlock.z - 1) / threadsPerBlock.z);
    convolution2D<<<blocksPerGrid, threadsPerBlock>>>(
            d_input, d_output, height, width, channels, d_kernel, kernelSize, padSize
    );

    hipDeviceSynchronize();

    hipMemcpy(output.ptr(), d_output, output.total() * output.elemSize(), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

int main() {
    // Read the input image using OpenCV
    cv::Mat originalImage = cv::imread("input.png");
    cv::resize(originalImage, originalImage, cv::Size(1000, 563));;

    if (originalImage.empty()) {
        std::cerr << "Error: Could not read the input image." << std::endl;
        return -1;
    }

    // Set the number of frames and frame rate
    int numFrames = 100;
    int frameRate = 10;  // frames per second

    // Create a VideoWriter object
    cv::VideoWriter videoWriter("blurred_video.avi", cv::VideoWriter::fourcc('X', 'V', 'I', 'D'), frameRate,
                                cv::Size(originalImage.cols, originalImage.rows));

    // Generate frames with varying blur intensity
    for (int i = 70; i < numFrames; ++i) {
//    for (int i = 0; i < numFrames; ++i) {
        auto start = std::chrono::high_resolution_clock::now();

        // Vary the blur intensity from 0 to 100
        float blur_intensity = static_cast<float>(i * 100) / (numFrames - 1);

        // Interpolate between [0, 100] to get the maximum blur size [1, 31]
        int max_blur_size = static_cast<int>(std::round((blur_intensity - 0) * (31 - 1) / (100 - 0) + 1));

        // Interpolate between [0, 100] to get the maximum blur strength [0.1, 10.0]
        float max_blur_strength = (blur_intensity - 0) * (10.0 - 0.1) / (100 - 0) + 0.1;

        // Output the values for the current frame

        // Apply Gaussian blur using CUDA
        cv::Mat frameBlurred(originalImage.size(), originalImage.type());
        applyGaussianBlurCUDA(originalImage, frameBlurred, max_blur_size, max_blur_strength);

//        if (i == 99) {
//            std::cout << "Frame index " << i << ": Blur Size = " << max_blur_size << ", Blur Strength = "
//                      << max_blur_strength << std::endl;
//        }

        // Write the frame to the video
        videoWriter.write(frameBlurred);

        // Print the frame generation time
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> duration = end - start;
        std::cout << "Frame " << i + 1 << "/" << numFrames << " - Time: " << duration.count() << " seconds"
                  << std::endl;
    }

    // Release the VideoWriter object
    videoWriter.release();

    std::cout << "Video created successfully." << std::endl;

    return 0;
}
