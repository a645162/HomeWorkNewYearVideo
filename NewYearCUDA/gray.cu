#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>

#define CONVERT_TO_GRAY_AVG(r, g, b) (uchar)((r + g + b) / 3)

#define WEIGHTED_GRAY_WEIGHT_R 0.299
#define WEIGHTED_GRAY_WEIGHT_G 0.587
#define WEIGHTED_GRAY_WEIGHT_B 0.114

#define CONVERT_TO_WEIGHTED_GRAY(r, g, b) (uchar)( \
    WEIGHTED_GRAY_WEIGHT_R * (r) + WEIGHTED_GRAY_WEIGHT_G * (g) + WEIGHTED_GRAY_WEIGHT_B * (b) \
)


// 0: average 1: weighted
__global__ void convertToGrayRGB(
        const uchar *inputImage, uchar *outputImage,
        int width, int height, int channels,
        int type = 0
) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        auto index = y * width + x;

        // Calculate pixel index for each channel
        auto blueIndex = index * channels;
        auto greenIndex = blueIndex + 1;
        auto redIndex = blueIndex + 2;
        auto alphaIndex = blueIndex + 3;

        auto r = inputImage[redIndex];
        auto g = inputImage[greenIndex];
        auto b = inputImage[blueIndex];

        uchar grayValue;

        if (type == 0) {
            grayValue = CONVERT_TO_GRAY_AVG(r, g, b);
        } else {
            grayValue = CONVERT_TO_WEIGHTED_GRAY(r, g, b);
        }

        if (channels == 4) {
            outputImage[alphaIndex] = inputImage[alphaIndex];
        } else if (channels != 3) {
            grayValue = 0;
        }

        // Set the same grayscale value for all channels
        outputImage[redIndex] = grayValue;
        outputImage[greenIndex] = grayValue;
        outputImage[blueIndex] = grayValue;
    }
}

__global__ void convertToGraySingleChannel(
        const uchar *inputImage, uchar *outputImage,
        int width, int height, int channels,
        int type = 0
) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        auto targetIndex = y * width + x;

        // Calculate pixel index for each channel
        auto blueIndex = targetIndex * channels;
        auto greenIndex = blueIndex + 1;
        auto redIndex = blueIndex + 2;

        auto r = inputImage[redIndex];
        auto g = inputImage[greenIndex];
        auto b = inputImage[blueIndex];

        uchar grayValue;

        if (type == 0) {
            grayValue = CONVERT_TO_GRAY_AVG(r, g, b);
        } else {
            grayValue = CONVERT_TO_WEIGHTED_GRAY(r, g, b);
        }

        outputImage[targetIndex] = grayValue;
    }
}

int main() {
    // Read the image using OpenCV
    cv::Mat inputImage = cv::imread("../Resources/input.png", cv::IMREAD_UNCHANGED);
    cv::resize(
            inputImage, inputImage,
            cv::Size(inputImage.cols / 4, inputImage.rows / 4)
    );

    if (inputImage.empty()) {
        std::cerr << "Failed to read the image." << std::endl;
        return -1;
    }

    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels(); // Get the number of channels

    std::cout << "channels: " << channels << std::endl;

    // Allocate memory on the host
    uchar *hostInputImage = inputImage.data;
    uchar *hostOutputImage = new uchar[inputImage.total() * channels];
    uchar *hostOutputImage1 = new uchar[inputImage.total() * channels];

    // Allocate memory on the device
    uchar *deviceInputImage;
    uchar *deviceOutputImage;
    uchar *deviceOutput1ChannelImage;

    hipMalloc((void **) &deviceInputImage, inputImage.total() * channels * sizeof(uchar));
    hipMalloc((void **) &deviceOutputImage, inputImage.total() * channels * sizeof(uchar));

    hipMalloc((void **) &deviceOutput1ChannelImage, inputImage.total() * sizeof(uchar));

    // Copy the input image to the device
    hipMemcpy(deviceInputImage, hostInputImage, inputImage.total() * channels * sizeof(uchar), hipMemcpyHostToDevice);

    // Specify block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the CUDA kernel
//    convertToGrayRGB<<<gridSize, blockSize>>>(
//            deviceInputImage, deviceOutputImage,
//            width, height, channels,
//            1
//    );

    convertToGrayRGB<<<gridSize, blockSize>>>(
            deviceInputImage, deviceOutputImage,
            width, height, channels,
            1
    );

    convertToGraySingleChannel<<<gridSize, blockSize>>>(
            deviceInputImage, deviceOutput1ChannelImage,
            width, height, channels,
            1
    );

    // Copy the result back to the host
    hipMemcpy(hostOutputImage, deviceOutputImage, inputImage.total() * channels * sizeof(uchar),
               hipMemcpyDeviceToHost);
//
//    // Create a new OpenCV image with the grayscale data
    cv::Mat outputImage(height, width, CV_8UC(channels), hostOutputImage);

    hipMemcpy(hostOutputImage1, deviceOutput1ChannelImage, inputImage.total() * sizeof(uchar),
               hipMemcpyDeviceToHost);
    cv::Mat outputImage1(height, width, CV_8UC1, hostOutputImage1);

    // Display the original and grayscale images
    cv::imshow("Original Image", inputImage);
    cv::imshow("Grayscale Image", outputImage);
    cv::imshow("Grayscale Image1", outputImage1);
    cv::waitKey(0);

    // Free memory
    delete[] hostOutputImage;
    hipFree(deviceInputImage);
    hipFree(deviceOutputImage);
}