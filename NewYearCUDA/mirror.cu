#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for horizontal mirroring
__global__ void robst(const uchar3* inputImage, uchar3* outputImage, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        // Mirror horizontally
        outputImage[y * width + x] = inputImage[y * width + (width - 1 - x)];
    }
}

int main() {
    // Read the image using OpenCV
    cv::Mat inputImage = cv::imread("image1.png");

    if (inputImage.empty()) {
        std::cerr << "Failed to read the image." << std::endl;
        return -1;
    }

    int width = inputImage.cols;
    int height = inputImage.rows;

    // Allocate memory on the host
    uchar3* hostInputImage = (uchar3*)inputImage.ptr();
    uchar3* hostOutputImage = new uchar3[width * height];

    // Allocate memory on the device
    uchar3* deviceInputImage;
    uchar3* deviceOutputImage;

    hipMalloc((void**)&deviceInputImage, width * height * sizeof(uchar3));
    hipMalloc((void**)&deviceOutputImage, width * height * sizeof(uchar3));

    // Copy the input image to the device
    hipMemcpy(deviceInputImage, hostInputImage, width * height * sizeof(uchar3), hipMemcpyHostToDevice);

    // Specify block and grid dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the CUDA kernel
    mirrorImage<<<gridSize, blockSize>>>(deviceInputImage, deviceOutputImage, width, height);

    // Copy the result back to the host
    hipMemcpy(hostOutputImage, deviceOutputImage, width * height * sizeof(uchar3), hipMemcpyDeviceToHost);

    // Create a new OpenCV image with the mirrored data
    cv::Mat outputImage(height, width, CV_8UC3, hostOutputImage);

    // Display the original and mirrored images
    cv::imshow("Original Image", inputImage);
    cv::imshow("Mirrored Image", outputImage);
    cv::waitKey(0);

    // Free memory
    delete[] hostOutputImage;
    hipFree(deviceInputImage);
    hipFree(deviceOutputImage);

    return 0;
}
